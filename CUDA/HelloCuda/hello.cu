#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
  printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main()
{
  hello<<<1,10>>>();
  hipDeviceSynchronize();
  printf("Hello from CPU\n");
}
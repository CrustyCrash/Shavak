#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void gpu_print(void)
{
  printf("Hello from GPU\n");
}

void cpu_print(void)
{
  printf("Hello from CPU\n");
}

int main()
{
  gpu_print<<<1,1>>>();
  gpu_print<<<1,1>>>();
  gpu_print<<<1,1>>>();

  hipDeviceSynchronize();

  cpu_print();
  cpu_print();
  cpu_print();

  return 0;
}
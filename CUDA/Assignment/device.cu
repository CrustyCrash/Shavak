#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int square(int num)
{
    // printf("Thread (%d, %d) - squaring value \n", blockIdx.x, threadIdx.x);
    return num * num;
}

__global__ void doubleValues(int* data, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
    {
        int value = data[i];
        int squared_value = square(value);
        // printf("Thread (%d, %d) - doubling squared value \n", blockIdx.x, threadIdx.x);
        data[i] = squared_value*2;
    }
}

int main()
{
    int size = 1000000;
    int* data_host = new int[size];
    int* data_device;
    hipMalloc(&data_device, size * sizeof(int));

    for(int i = 0; i < size; i++)
    {
        data_host[i] = i;
    }
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);

    int threadPerBlock = 1024;
    int blockPerGrid = (size + threadPerBlock - 1) / threadPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    doubleValues<<<blockPerGrid, threadPerBlock>>>(data_device, size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(data_host, data_device, size*sizeof(int), hipMemcpyDeviceToHost);
    // for(int i = 0; i < size; i++)
    // {
    //     printf("%d ", data_host[i]);
        
    // }
    printf("Time taken to execute kernel function: %fms\n", ms);
    hipFree(data_device);
    delete[] data_host;
    return 0;
}
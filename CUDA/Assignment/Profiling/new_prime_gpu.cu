#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include <math.h>

#define N 100
int threads = 100;
// Calculating number of blocks
int blocks = (N + threads - 1) / threads;

__device__ bool isprime(int tid)
{
    if(tid < 2) return false;
    if (tid % 2 == 0) return false;
    if (tid == 2) return true;
 
    for (int i = 3; i <= sqrtf((float)tid); i += 2)
    {
        if (tid % i == 0) return false;
    }
    return true;
}

__global__ void prime(int* count)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= 2 && tid < N && isprime(tid))
    {
        atomicAdd(count, 1);
    }
}

int main()
{
    int* count;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&count, sizeof(int));
    hipMemset(count, 0, sizeof(int));

    
    hipEventRecord(start);
    prime<<<blocks, threads>>>(count);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    int result;
    hipMemcpy(&result, count, sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Number of primes between 2 and %d: %d\n", N, result);
    printf("Time taken: %f ms\n", milliseconds);

    hipFree(count);
    return 0;
}
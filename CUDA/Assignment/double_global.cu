#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int size = 1000000;
int threadPerBlock = 1024;
int blockPerGrid = (size + threadPerBlock - 1) / threadPerBlock;

__global__ void square(int* num)
{
    int value = *num;
    *num = value * value;
}

__global__ void doubleValues(int* data, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
    {
        int value = data[i];
        int* d_value;
        hipMalloc((void**)&d_value, sizeof(int));
        hipMemcpy(d_value, &value, sizeof(int), hipMemcpyHostToDevice);

        // Launch square kernel dynamically
        square<<<1, 1>>>(d_value);
        hipDeviceSynchronize(); // Wait for the square kernel to complete

        hipMemcpy(&value, d_value, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_value);

        data[i] = value * 2;
    }
}

int main()
{
    int* data_host = new int[size];
    int* data_device;
    hipMalloc(&data_device, size * sizeof(int));

    for (int i = 0; i < size; i++)
    {
        data_host[i] = i;
    }
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    doubleValues<<<blockPerGrid, threadPerBlock>>>(data_device, size);
    hipDeviceSynchronize(); // Wait for the doubleValues kernel to complete

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(data_host, data_device, size * sizeof(int), hipMemcpyDeviceToHost);

    printf("Time taken to execute kernel function: %fms\n", ms);
    
    hipFree(data_device);
    delete[] data_host;
    return 0;
}
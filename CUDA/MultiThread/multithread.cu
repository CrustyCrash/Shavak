#include <stdio.h>
#include <hip/hip_runtime.h>

// kernel to be executed on gpu
__global__ void gpu_print(void)
{
  printf("Block ID: %d, Thread ID: %d\n", blockIdx.x, threadIdx.x);
  printf("Global ID: %d\n", threadIdx.x + blockIdx.x * blockDim.x);
}

void cpu_print(void)
{
  printf("Hello from CPU\n");
}

int main()
{
  gpu_print<<<2,10>>>();
  hipDeviceSynchronize();

  cpu_print();
  
  return 0;
}
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void doubleValues(int* data, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
    {
       data[i] = 2 * (data[i] * data[i]);
    }
}

int main()
{
    int size = 1000000;
    int* data_host = new int[size];
    int* data_device;
    hipMalloc(&data_device, size * sizeof(int));

    for(int i = 0; i < size; i++)
    {
        data_host[i] = i;
    }
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);

    int threadPerBlock = 1024;
    int blockPerGrid = (size + threadPerBlock - 1) / threadPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    doubleValues<<<blockPerGrid, threadPerBlock>>>(data_device, size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(data_host, data_device, size*sizeof(int), hipMemcpyDeviceToHost);
    // for(int i = 0; i < size; i++)
    // {
    //     printf("%d ", data_host[i]);
        
    // }
    printf("Time taken to execute kernel function: %fms\n", ms);
    hipFree(data_device);
    delete[] data_host;
    return 0;
}
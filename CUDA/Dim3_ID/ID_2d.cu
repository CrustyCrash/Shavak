#include "hip/hip_runtime.h"
#include <stdio.h>
#include
#include <hip/hip_runtime.h>
#include<stdlib.h>

#define N 4
int rows = 10;
int cols = 10;
dim3 dimBlock(N,N);
dim3 dimGrid(rows + N - 1 / N, cols + N - 1 / N);

__global__ void displayGlobalId()
{
    int tidX = threadIdx.x + blockIdx.x * blockDim.x;
    int tidY = threadIdx.y + blockIdx.y * blockDim.y;
    int tBlocks = N*N;
    int Gid = tidX * tBlocks + tidY;
    printf("Global TID: %d\n", Gid);
}

int main()
{
    displayGlobalId<<<dimGrid,dimBlock>>>();
    hipDeviceSynchronize();
    return 0;
}

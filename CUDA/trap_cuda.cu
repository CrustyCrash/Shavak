#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ double f(double x)
{
    return 1/(1+x*x);
}

__global__ void trapezoidal(double a, double b, int n)
{
    double h = (b-a)/n;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid > 1 && tid < n)
    {
    double x = a + tid * h;
    double my_trap = f(x);
    atomicAdd(sum, my_trap);
    }
}

int main()
{
    double a = 0.0;
    double b = 1.0;
    double* sum;
    int n = 100000;
    hipMallocManaged(&sum, sizeof(double));

    *sum = 0.5(f(a) + f(b));
    

}


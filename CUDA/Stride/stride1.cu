#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1000

__global__ void vector_add(int* dev_a, int* dev_b, int* dev_c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = tid; i < N; i+=blockDim.x)
    {
        dev_c[i] = dev_a[i] + dev_b[i];
    }
}

int main()
{
    int* a = (int*)malloc(N * sizeof(int));
    int* b = (int*)malloc(N * sizeof(int));;
    int* c = (int*)malloc(N * sizeof(int));;
    int *dev_a, *dev_b, *dev_c;

    for(int i = 1; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    hipMemcpy(dev_a,a,N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(int), hipMemcpyHostToDevice);

    int block_size = 1;
    int thread_num = 100;

    vector_add<<<block_size,thread_num>>>(dev_a,dev_b,dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    bool flag = true;
    for(int i = 0; i < N; i++)
    {

        if(a[i] + b[i] != c[i])
        {
            printf("Error at %d + %d != %d\n",a[i],b[i],c[i]);
            flag = false;
        }
    }
    if(flag)
    {
        printf("Program executed successfully!\n");
    }
    free(a);
    free(b);
    free(c);
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
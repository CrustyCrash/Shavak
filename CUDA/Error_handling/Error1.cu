#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

const int size = 100;
__global__ void kernel(int* dev_a, int* dev_b, int* dev_c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // int blockID = blockIdx.x;
    // int threadID = threadIdx.x;
    // int stride = blockDim.x * gridDim.x;
    // int threadPerBlock = blockDim.x;

    if(tid < size)
    {
        dev_c[tid] = dev_a[tid] + dev_b[tid];
    }
}


int main()
{
    
    srand(time(0));
    int hostarray[size];
    int hostarray2[size];
    int hostresult[size];

    for(int i = 0 ; i < size; i++)
    {
        hostarray[i] = rand() % 100;
        hostarray2[i] = rand() % 100;

    } 

    //GPU data
    int* deviceArray1;
    int* deviceArray2;
    int* deviceResultArray;

    //allocate device memory
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&deviceArray1, size*sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed for deviceArray1! : %s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__,__LINE__);
        hipFree(deviceArray1);
        exit(EXIT_FAILURE);
    }
    cudaStatus = hipMalloc((void**)&deviceArray2, size*sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed for deviceArray2! :%s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__,__LINE__);
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        exit(EXIT_FAILURE);
    }
    cudaStatus = hipMalloc((void**)&deviceResultArray, size*sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed for deviceResultArray! : %s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__,__LINE__);
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }
    //copy host data to device
    cudaStatus = hipMemcpy(deviceArray1, hostarray, size*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed for deviceArray1! :  %s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__,__LINE__);
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }
    cudaStatus = hipMemcpy(deviceArray2, hostarray2, size*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed for deviceArray2! :  %s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__,__LINE__);
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }


    //launching the kernel
    kernel<<<1,size>>>(deviceArray1,deviceArray2,deviceResultArray);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }
    //copying the result from device to host
    cudaStatus = hipMemcpy(hostresult, deviceResultArray, size*sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr,"Failed MemcpyDeviceToHost:  %s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__,__LINE__);
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }
    //printing the result
    for (int i = 0; i < size; i++)
    {
        printf("%d + %d = %d \n",hostarray[i], hostarray2[i], hostresult[i]);
    }
    printf("\n");
    //freeing the memory
    hipFree(deviceArray1);
    hipFree(deviceArray2);
    hipFree(deviceResultArray);
}
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N (33 * 1024)

//kernel to be excuted on GPU
__global__ void sum(int* dev_a, int* dev_b, int* dev_c)
{
    // global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    //only making the threads with index < N perform the action to prevent out of bound errors
    if (tid < N)
    {
        dev_c[tid] = dev_a[tid] + dev_b[tid];
    }
}

int main()
{
    int* a = (int*)malloc(N * sizeof(int));
    int* b = (int*)malloc(N * sizeof(int));
    int* c = (int*)malloc(N * sizeof(int));

    //initialising the array on the cpu
    for(int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = -i;
    }

    // pointers on device
    int* dev_a;
    int* dev_b;
    int* dev_c;

    //allocating memory on the device
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    //copying array from host to device
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    //calling kernel with 512 blocks and 512 threads
    sum <<<512,512>>>(dev_a, dev_b, dev_c);
    
    //copying result from device to host
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    bool verify = true;

    //verifying if the output is correct
    for(int i = 0; i < N; i++)
    {
        if(a[i] + b[i] != c[i])
        {
            printf("Failed at %d + %d != %d\n",a[i],b[i],c[i]);
            verify = false;
        }
    }
    
    if(verify)
    {
        printf("Program executed successfully!\n");
    }

    free(a);
    free(b);
    free(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
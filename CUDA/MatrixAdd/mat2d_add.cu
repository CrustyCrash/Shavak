#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 16

__global__ void add2D(int* a, int* b, int* c, int rows, int cols)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

   if(i < rows && j < cols) 
    {
        c[i*cols + j] = a[i*cols + j] + b[i*cols + j];
    } 
}

int main()
{
    int rows = 4096;
    int cols = 2048;
    dim3 dimBlock(N,N);
    dim3 dimGrid ((rows+N-1)/N, (cols+N-1)/N);
    int *a = (int*)malloc(rows * cols * sizeof(int));
    int *b = (int*)malloc(rows * cols * sizeof(int));
    int *c = (int*)malloc(rows * cols * sizeof(int));

    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipError_t err = hipMalloc((void**)&dev_a, rows*cols*sizeof(int));
    if(err!= hipSuccess)
    {
        printf("Error: %s in file %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&dev_b, rows*cols*sizeof(int));
    if(err!= hipSuccess)
    {
        printf("Error: %s in file %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&dev_c, rows*cols*sizeof(int));
    if(err!= hipSuccess)
    {
        printf("Error: %s in file %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i < rows; i++)
    {
        for(int j = 0; j < cols; j++)
        {
            a[i*cols+j] = i+i;
            b[i*cols+j] = i+j;
        }
    }

    hipError_t memCheck = hipMemcpy(dev_a,a,rows*cols*sizeof(int),hipMemcpyHostToDevice);
    if(memCheck!= hipSuccess)
    {
        printf("Error: %s in file %s at line %d\n", hipGetErrorString(memCheck), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    memCheck = hipMemcpy(dev_b,b,rows*cols*sizeof(int),hipMemcpyHostToDevice);
    if(memCheck!= hipSuccess)
    {
        printf("Error: %s in file %s at line %d\n", hipGetErrorString(memCheck), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    add2D<<<dimGrid,dimBlock>>>(dev_a, dev_b, dev_c, rows, cols);

    memCheck = hipMemcpy(c,dev_c,rows*cols*sizeof(int),hipMemcpyDeviceToHost);
    if(memCheck!= hipSuccess)
    {
        printf("Error: %s in file %s at line %d\n", hipGetErrorString(memCheck), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    bool flag = true;
    for(int i = 0; i < rows; i++)
    {
        for(int j = 0; j < cols; j++)
        {
            if(c[i*cols+j]!=a[i*cols+j]+b[i*cols+j])
            {
                printf("Error! %d + %d != %d ", a[i*cols+j], b[i*cols+j], c[i*cols+j] );
                flag = false;
            }

        }
    }
    if(flag)
    {
        printf("Program executed successfully!");
    }

    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define N 100001

int main()
{
    int count = 0;
    
    for(int i = 2; i < N+1; i++)
    {
        if (i==2) 
        {
            count++;
            continue;
        }
        if(i%2==0)
        {
            continue;
        }

        bool flag = true;
        
        for(int j = 3; j <= sqrt(i); j+=2 )
        {
            if(i%j==0)
            {
                flag = false;
                break;
            }
        }
        if(flag)
        {
            count++;
        }   

    }
    printf("%d\n",count);

}
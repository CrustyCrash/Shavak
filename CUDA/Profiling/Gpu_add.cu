#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 90000
int threads = 1024;
//calculating number of blocks
int block = (N+threads-1)/threads;

__global__ void add(int* dev_a, int* dev_b, int* dev_c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N)
    {
        dev_c[tid] = dev_a[tid] + dev_b[tid];
    }

}

int main()
{
    int* a = (int*)malloc(N * sizeof(int));
    int* b = (int*)malloc(N * sizeof(int));
    int* c = (int*)malloc(N * sizeof(int));


    for(int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    //start timing GPU execution 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    add <<<block,threads>>> (dev_a, dev_b, dev_c);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    bool verify = true;
    for(int i = 0; i < N; i++)
    {
        if(a[i] + b[i] != c[i])
        {
            printf("Failed at %d + %d != %d\n",a[i],b[i],c[i]);
            verify = false;
        }
    }
    
    if(verify)
    {
        printf("Program executed successfully!\n");
    }

    printf("Time taken by GPU: %fms\n", ms);

    free(a);
    free(b);
    free(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

}

